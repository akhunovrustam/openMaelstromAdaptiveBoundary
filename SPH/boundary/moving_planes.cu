#include "hip/hip_runtime.h"
#define NEW_STYLE
#include <SPH/boundary/moving_planes.cuh>
#include <utility/include_all.h>

// This function adjusts the velocity of a particle against a moving boundary by simply removing the
// velocity of the particle in direction of the boundary. This function is called once per
// boundary described by an implicit plane equation E(n.x, n.y, n.z, d) and the velocity of the
// boundary.
basicFunctionType correct_velocity_moving(SPH::moving_planes::Memory arrays, float4_u<void_unit_ty> E, float4_u<SI::velocity>) {
  checkedParticleIdx(i);

  auto r = arrays.radius;
  auto p = arrays.position.first[i];
  auto v = arrays.velocity.first[i];
  auto dt = arrays.timestep;

  float4_u<> n{ E.val.x, E.val.y, E.val.z, 0.f };
  n = math::normalize(n);

  auto pe = p + v * dt;
  auto dist = math::planeDistance(E, pe);

  if (dist < r) {
    auto vTn = math::dot3(n, v) * n;
    auto v2 = v - vTn;
    v = v2;
  }
  arrays.velocity.first[i] = v;
}

// This function adjusts the position of a particle against a moving boundary by simply reprojecting
// the particle away from the wall. This process usually causes some errors, usually visible as
// pressure fluctuations on the boundary, but due to these walls possibly being infinite and planar
// representing them with particles could become very expensive. Additionally using implicit planes
// to describe the boundaries makes this process very quick. This function is called once per
// boundary described by an implicit plane equation E(n.x, n.y, n.z, d) and the velocity of the
// boundary.
basicFunctionType correct_position_moving(SPH::moving_planes::Memory arrays, float4_u<void_unit_ty> E, float4_u<SI::velocity> v_diff) {
  checkedParticleIdx(i);
  auto r = arrays.radius;
  auto p = arrays.position.first[i];
  auto v = arrays.velocity.first[i];
  auto dt = arrays.timestep;
  float4_u<> n{E.val.x, E.val.y, E.val.z, 0.f};
  n = math::normalize(n);

  auto d = math::planeDistance(E, p);
  if (d < r)
    p += (r - d) * n;

  auto pe = p + v * dt;
  d = math::planeDistance(E, pe);

  bool hit = false;
  if (d < r) {
    v = v - math::dot3(n, v) * n;
    hit = true;
  }
  if (hit)
    v += v_diff;
  v.val.w = 0.f;

  p -= arrays.velocity.first[i] * dt;
  arrays.velocity.first[i] = v;
  pe = p + dt * v;
  d = math::planeDistance(E, pe);
  if (d < r)
    p += (r - d) * n;
  arrays.position.first[i] = p;
}

// Launcher to correct the velocity of particles against a single boundary
basicFunction(correctVelocity, correct_velocity_moving, "Moving Planes: correct velocity");
// Launcher to correct the position of particles against a single boundary
basicFunction(correctPosition,correct_position_moving, "Moving Planes: correct position");
// Launcher to correct the position of particles against a single boundary
basicFunctionType update_plane_moving(int32_t threads, SPH::moving_planes::Memory arrays, float4_u<void_unit_ty> E, float4_u<SI::velocity> v, int32_t idx) {
  checkedThreadIdx(i);
  arrays.boundaryPlanes[idx] = E;
  arrays.boundaryPlaneVelocity[idx] = v;
}
basicFunction(updatePlane,update_plane_moving, "Moving Planes: update planes");


// Main entry function of the module for correcting positions. Iterates over all boundaries and
// calls the appropriate correction functions once for each boundary. This method can be called
// safely if no boundaries exist.
void SPH::moving_planes::correct_position(Memory mem) {
  for (auto plane : get<parameters::moving_plane::plane>()) {
    float t     = plane.duration;
    float f     = plane.frequency;
    float m     = plane.magnitude;
    float3 p    = plane.plane_position;
    float3 n    = plane.plane_normal;
    float3 dir  = plane.plane_direction;

    p          += dir * m * sinf(2.f * HIP_PI_F * f * (get<parameters::internal::simulationTime>()));

    auto p_prev = dir * m * sinf(2.f * HIP_PI_F * f * (get<parameters::internal::simulationTime>() - get<parameters::internal::timestep>()));
    auto p_diff = p - p_prev;
    auto v_diff = p_diff / get<parameters::internal::timestep>();

    auto nn     = math::normalize(n);
    auto d      = math::dot3(p, nn);
    float4_u<> E{nn.x, nn.y, nn.z, d};

    if (t < get<parameters::internal::simulationTime>() && t > 0.f)
      continue;
    launch<correctVelocity>(mem.num_ptcls, mem, E, float4_u<SI::velocity>{v_diff.x, v_diff.y, v_diff.z, 0.f});
  }
}

// Main entry function of the module for correcting velocities. Iterates over all boundaries and
// calls the appropriate correction functions once for each boundary. This method can be called
// safely if no boundaries exist.
void SPH::moving_planes::correct_velocity(Memory mem) {
  for (auto plane : get<parameters::moving_plane::plane>()) {
    auto t = plane.duration;
    auto f = plane.frequency;
    auto m = plane.magnitude;
    auto p = plane.plane_position;
    auto n = plane.plane_normal;
    auto dir = plane.plane_direction;
    p += dir * m * sinf(2.f * HIP_PI_F * f * get<parameters::internal::simulationTime>());

    auto p_prev = plane.plane_position + dir * m * sinf(2.f * HIP_PI_F * f * (get<parameters::internal::simulationTime>() - get<parameters::internal::timestep>()));
    auto p_diff = p - p_prev;
    auto v_diff = -p_diff / get<parameters::internal::timestep>();

    auto nn     = math::normalize(n);
    auto d      = math::dot3(p, nn);
    float4_u<> E{nn.x, nn.y, nn.z, d};

    if (t < get<parameters::internal::simulationTime>() && t > 0.f)
      continue;
    launch<correctPosition>(mem.num_ptcls, mem, E, float4_u<SI::velocity>{v_diff.x, v_diff.y, v_diff.z, 0.f});
  }
}

void SPH::moving_planes::update_boundaries(Memory mem){
  for (auto plane : get<parameters::moving_plane::plane>()) {
    auto t = plane.duration;
    auto f = plane.frequency;
    auto m = plane.magnitude;
    auto p = plane.plane_position;
    auto n = plane.plane_normal;
    auto dir = plane.plane_direction;
    auto idx = plane.index;
    //if (get<parameters::internal::simulationTime>() > 30.f) return;
    p += dir * m * sinf(2.f * HIP_PI_F * f * get<parameters::internal::simulationTime>());
     
    if (math::dot3(n, float3{ 1,0,0 }) > 0.5f)
        get<parameters::render_settings::vrtxRenderDomainMin>().x = p.x;
    if (math::dot3(n, float3{ 1,0,0 }) < -0.5f)
        get<parameters::render_settings::vrtxRenderDomainMax>().x = -p.x;
    if (math::dot3(n, float3{ 0,1,0 }) > 0.5f)
        get<parameters::render_settings::vrtxRenderDomainMin>().y = p.y;
    if (math::dot3(n, float3{ 0,1,0 }) < -0.5f)
        get<parameters::render_settings::vrtxRenderDomainMax>().y = -p.y;
    if (math::dot3(n, float3{ 0,0,1 }) > 0.5f)
        get<parameters::render_settings::vrtxRenderDomainMin>().z = p.z;
    if (math::dot3(n, float3{ 0,0,1 }) < -0.5f)
        get<parameters::render_settings::vrtxRenderDomainMax>().z = -p.z;

    auto p_prev = plane.plane_position + dir * m * sinf(2.f * HIP_PI_F * f * (get<parameters::internal::simulationTime>() - get<parameters::internal::timestep>()));
    auto p_diff = p - p_prev;
    auto v_diff = -p_diff / get<parameters::internal::timestep>();

    auto nn     = math::normalize(n);
    auto d      = math::dot3(p, nn);
    float4_u<> E{nn.x, nn.y, nn.z, d};

    if (t < get<parameters::internal::simulationTime>() && t > 0.f)
      continue;
	uFloat4<SI::velocity> v{ math::castTo<float4>(v_diff) };
    launch<updatePlane>(1, 1, mem, E, v, idx);
  }

}